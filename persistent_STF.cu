#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<algorithm>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h>


//#define PSIZE 10
//#define NGEN 500000
#define MUT_PROB 0.05
#define TESTE 256
#define CHROMO_SIZE 256

struct Individual 
{
    float fitness;
    char chromossomes[CHROMO_SIZE];
};

__device__ bool comparator (Individual i, Individual j)
{
    return (i.fitness > j.fitness);
}

void printPop(Individual *population, int popSize, int print)
{
    if(print != 0)
    {
        for(int i = 0; i < popSize; i++)
        {
            printf("%f - ", population[i].fitness);
        }
    }
}

/*__device__ float fitness(Individual *population, int id)
{
    unsigned int mask = 0x3FF;
    float a = 0, b = 0, c = 0;
    a = population[id].chromossomes & mask;
    b = (population[id].chromossomes & (mask << 9)) >> 9;
    c = (population[id].chromossomes & (mask << 18)) >> 18;

    a = (a - 512)/100.0;
    b = (b - 512)/100.0;
    c = (c - 512)/100.0;

    population[id].fitness = 1.0 / (1 + a*a + b*b + c*c);
}*/




//__global__ void persistentThreads(Individual *population, Individual *nextPopulation, int popSize, int NGEN, float *maxFitness, unsigned int seed)
__global__ void persistentThreads(int popSize, int NGEN, float *maxFitness, unsigned int seed)
{
 //   printf("KERNEL\n");
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    Individual child;
    hiprandState_t state;
    hiprand_init(seed, id, 0, &state);
 //   hiprand_init(0, id, 0, &state);
    //numbers[id] = hiprand(&state) % 100;

  //  Individual *swap;
//    population = pop;
  //  nextPopulation = nextPop;
    __shared__ float totalFitness;
    extern __shared__ Individual population[];

    //printf("%d ========== ", popSize);

    for(int i = id; i < popSize; i+=blockDim.x)
    {
        //Create population
        population[i].fitness = 0;
        for(int j = 0; j < CHROMO_SIZE; j++)
        {
            population[i].chromossomes[j] = hiprand(&state)%256 - 128;
        }
        //printf("%x - ", population[i].chromossomes);
    }
    __syncthreads();

    /*if(id == 0)
    {  
        printf("LALAAL\n");
        for(int i = 0; i < popSize; i++)
        {
            printf("%f ; %x\n", population[i].fitness, population[i].chromossomes);
        }
        printf("LALAAL\n");
    }*/

    for(int g = 0; g < NGEN; g++)
    {
        if(id == 0)
        {
            totalFitness = 0;
        }
        __syncthreads();
        for(int i = id; i < popSize; i+=blockDim.x)
        {
            //Calculate fitness

            float subTotal = 0;
            for(int j = 0; j < CHROMO_SIZE; j++)
            {
                float x = (population[i].chromossomes[j] / 128.0) * 5.0;
                subTotal += (x*x*x*x - 16.0*x*x + 5.0*x) / 2.0;
            }
            population[i].fitness = 1.0 / (40.0*CHROMO_SIZE + subTotal);

            atomicAdd(&totalFitness, population[i].fitness);
        }
        __syncthreads();
        if(id == 0)
        {
            thrust::sort(population, population + popSize, comparator);
            maxFitness[g] = population[0].fitness;
 //           (*nextPopulation)[0] = (*population)[0];
        }
        __syncthreads();
        
        float localTotalFitness = totalFitness;

        for(int i = id; i < popSize; i+=blockDim.x)
        {

            Individual parents[2];
            int temp = -1;

            //Selection
            for(int j = 0; j < 2; j++)
            {
                float p = hiprand_uniform(&state) * localTotalFitness;
                float score = 0;

                for(int k = 0; k < popSize; k++)
                {
                    if(k == temp)
                    {
                        continue;
                    }
                    score += population[k].fitness;
                    if(p < score)
                    {
                        parents[j] = population[k];
                        localTotalFitness -= population[k].fitness;
                        temp = k;
                        break;
                    }
                }
            }


                //Crossover
                unsigned char cutPoint = hiprand(&state) % (CHROMO_SIZE + 1);
                child.fitness = 0;
                for(int j = 0; j < cutPoint; j++)
                {
                    child.chromossomes[j] = parents[0].chromossomes[j];
                }

                for(int j = cutPoint; j < CHROMO_SIZE; j++)
                {
                    child.chromossomes[j] = parents[1].chromossomes[j];
                }

                //Mutation
                float mutation = hiprand_uniform(&state);
                if(mutation < MUT_PROB)
                {
                    int mutPoint = hiprand(&state) % CHROMO_SIZE;
                    child.chromossomes[mutPoint] = hiprand(&state) % 256 - 128;
                }

        }
        __syncthreads();
        
        if(id == 0)
        {
         //   printf("PA: %x\n", population[1].chromossomes);
           // printf("NPA: %x\n", nextPopulation[1].chromossomes);

           // nextPopulation[0] = population[0];
            child = population[0];
            //(*population) = NULL;

 //           printf("PD: %x\n", population[1].chromossomes);
   //         printf("NPD: %x\n", nextPopulation[1].chromossomes);
        }

        for(int i = id; i < popSize; i+=blockDim.x)
        {
            //population[i] = nextPopulation[i];
            population[i] = child;
        }
        __syncthreads();
        
    }
/*
    for(int iii = id; iii < popSize; iii+=blockDim.x)
    {
        //Calculate fitness
        
        unsigned int mask = 0x3FF;
        float a = 0, b = 0, c = 0;
        a = population[iii].chromossomes & mask;
        b = (population[iii].chromossomes & (mask << 9)) >> 9;
        c = (population[iii].chromossomes & (mask << 18)) >> 18;

        a = (a - 512)/100.0;
        b = (b - 512)/100.0;
        c = (c - 512)/100.0;

        population[iii].fitness = 1.0 / (1 + a*a + b*b + c*c);
        atomicAdd(&totalFitness, population[iii].fitness);
    }
    __syncthreads();
    if(id == 0)
    {
        thrust::sort(population, population + popSize, comparator);
//           (*nextPopulation)[0] = (*population)[0];
    }
*/
}

/*

int main()
{
  unsigned char *cpu_nums;
  cpu_nums = (unsigned char *) malloc(TESTE * sizeof(unsigned char));

  unsigned char* gpu_nums;
  hipMalloc((void**) &gpu_nums, TESTE * sizeof(unsigned char));

//  hiprandState *states;
 // hipMalloc((void**) &states, TESTE * sizeof(hiprandState_t));

  for(int i = 0; i < TESTE; i++)
  {
    cpu_nums[i] = 0;
    printf("%d - ", cpu_nums[i]);
  }
  persistentThreads<<<1, TESTE>>>(NULL, 0, 0, time(NULL), gpu_nums);

  hipError_t err;
  err = hipMemcpy(cpu_nums, gpu_nums, TESTE * sizeof(unsigned char), hipMemcpyDeviceToHost);

  printf("\n%u\n", err);
  for(int i = 0; i < TESTE; i++)
  {
    printf("%u - ", cpu_nums[i]);
  }


return 0;
}

*/


int main(int argc, char *argv[ ]) 
{
    int PSIZE, NGEN, NIT, PRINT;
    double Ttotal = 0;
    if(argc < 5)
    {
        printf("Uso %s <POP_SIZE> <N_GEN> <N_ITERACOES> <PRINT>\n", argv[0]);
        return 1;
    }
    else
    {
        PSIZE = atoi(argv[1]);
        NGEN = atoi(argv[2]);
        NIT = atoi(argv[3]);
        PRINT = atoi(argv[4]);
    }

    for(int it = 0; it < NIT; it++)
    {   
        clock_t start, end;
        start = clock();
        /*
        //Init variables
        Individual *population, *nextPopulation;
        hipMalloc((void**) &population, PSIZE * sizeof(Individual));
        hipMalloc((void**) &nextPopulation, PSIZE * sizeof(Individual));

        Individual *cpu_population, *cpu_nextPopulation;
        cpu_population = (Individual *) malloc(PSIZE * sizeof(Individual));
        cpu_nextPopulation = (Individual *) malloc(PSIZE * sizeof(Individual));

*/

        float *maxFitness, *cpu_maxFitness;
        hipMalloc((void**) &maxFitness, NGEN * sizeof(float));
        cpu_maxFitness = (float *) malloc(NGEN * sizeof(float));






        //persistentThreads<<<1, min(PSIZE, 1024), PSIZE * sizeof(Individual)>>>(PSIZE, NGEN, maxFitness, time(NULL));
 //       printf("%d\t%d", sizeof(Individual), PSIZE * sizeof(Individual));
        persistentThreads<<<1, min(PSIZE, 1024), PSIZE * sizeof(Individual)>>>(PSIZE, NGEN, maxFitness, time(NULL));

        //hipMemcpy(cpu_population, population, PSIZE * sizeof(Individual), hipMemcpyDeviceToHost);
        //hipMemcpy(cpu_nextPopulation, nextPopulation, PSIZE * sizeof(Individual), hipMemcpyDeviceToHost);
   //     printf("%s\n", hipGetErrorString(hipPeekAtLastError()));
        hipMemcpy(cpu_maxFitness, maxFitness, NGEN * sizeof(float), hipMemcpyDeviceToHost);
 //       printf("%s\n", hipGetErrorString(hipPeekAtLastError()));

/*
        for(int i = 0; i < PSIZE; i++)
        {
            printf("%f ; %x\n", cpu_population[i].fitness, cpu_population[i].chromossomes);
        }

        for(int i = 0; i < PSIZE; i++)
        {
            printf("%f ; %x\n", cpu_nextPopulation[i].fitness, cpu_nextPopulation[i].chromossomes);
        }
  */      
        end = clock();
        hipFree(maxFitness);
        if(PRINT != 0)
        {
            printf("Gen\tFitness\n");
            for(int i = 0; i < NGEN; i++)
            {
                printf("%d\t%f\n", i, cpu_maxFitness[i]);
            }
        }
        free(cpu_maxFitness);

        printf("\nT total(us)\t\tT geração(us)\n");
        double cpu_time_used = 1000000 * ((double) (end - start)) / CLOCKS_PER_SEC;
        printf("%f\t\t%f\n\n", cpu_time_used, cpu_time_used/NGEN);
        Ttotal += cpu_time_used;


        //free(cpu_population);

        //hipFree(population);
        //hipFree(nextPopulation);
    }

    printf("\nAvg T total(us)\t\tAvg T geração(us)\n");
    printf("%f\t\t%f\n", Ttotal/NIT, Ttotal/(NIT*NGEN));
    
return 0;
}

