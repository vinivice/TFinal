#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<algorithm>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h>


//#define PSIZE 10
//#define NGEN 500000
#define MUT_PROB 0.05

struct Individual 
{
    float fitness;
    unsigned int chromossomes;
};

bool comparator (Individual i, Individual j)
{
    return (i.fitness > j.fitness);
}

Individual individualSum (Individual i, Individual j)
{
    Individual I;
    I.fitness = i.fitness + j.fitness;
    return I;
}

void printPop(Individual *population, int popSize, int print)
{
    if(print != 0)
    {
        for(int i = 0; i < popSize; i++)
        {
            printf("%f - ", population[i].fitness);
        }
    }
}

__global__ void createPopulation(Individual *population, unsigned int seed, hiprandState_t *states)
{
    int id = blockIdx.x;
    hiprand_init(seed, id, 0, &states[id]);

    population[id].fitness = 0;
    population[id].chromossomes = hiprand(&states[id]);
}

__global__ void fitness(Individual *population)
{
    int id = blockIdx.x;
    unsigned int mask = 0x3FF;
    __shared__ float a[3];
        
    a[threadIdx.x] = (population[id].chromossomes & (mask << (9 * threadIdx.x))) >> (9 * threadIdx.x);

    a[threadIdx.x] = (a[threadIdx.x] - 512)/100.0;

    __syncthreads();

    if(threadIdx.x == 0)
    {
        population[id].fitness = 1.0 / (1 + a[0]*a[0] + a[1]*a[1] + a[2]*a[2]);
    }
}

__global__ void reproduce(Individual *population, Individual *nextPopulation, int PSIZE, float totalFitness, hiprandState_t *states)
{ 
    int id = blockIdx.x;
    Individual parents[2];
    int temp = -1;
    float localTotalFitness = totalFitness;
    //Selection
    for(int j = 0; j < 2; j++)
    {
        float p = hiprand_uniform(&states[id]) * localTotalFitness;
        float score = 0;

        for(int k = 0; k < PSIZE; k++)
        {
            if(k == temp)
            {
                continue;
            }
            score += population[k].fitness;
            if(p < score)
            {
                parents[j] = population[k];
                localTotalFitness -= population[k].fitness;
                temp = k;
                break;
            }
        }
    }

    //Crossover
    unsigned char cutPoint = hiprand(&states[id]) % 28;
    unsigned mask1 = 0xffffffff << cutPoint; 
    unsigned mask2 = 0xffffffff >> (32 - cutPoint);
    Individual child;
    child.fitness = 0;
    child.chromossomes = (parents[0].chromossomes & mask1) + (parents[1].chromossomes & mask2);

    //Mutation
    float mutation = hiprand_uniform(&states[id]);
    if(mutation < MUT_PROB)
    {
        unsigned char mutPoint = hiprand(&states[id]) % 27;
        child.chromossomes ^= 1 << mutPoint;
    }



    nextPopulation[id] = child;
    if(id == 0)
    {
        nextPopulation[0] = population[0];
    }


}




int main(int argc, char *argv[ ]) 
{
    int PSIZE, NGEN, NIT, PRINT;
    double Ttotal = 0;
    if(argc < 5)
    {
        printf("Uso %s <POP_SIZE> <N_GEN> <N_ITERACOES> <PRINT>\n", argv[0]);
        return 1;
    }
    else
    {
        PSIZE = atoi(argv[1]);
        NGEN = atoi(argv[2]);
        NIT = atoi(argv[3]);
        PRINT = atoi(argv[4]);
    }

    for(int it = 0; it < NIT; it++)
    {
        clock_t start, end;
        start = clock();
        //Init variables

        Individual *population, *nextPopulation, *swap;
        hipMalloc((void**) &population, PSIZE * sizeof(Individual));
        hipMalloc((void**) &nextPopulation, PSIZE * sizeof(Individual));


        hiprandState_t *states;
        hipMalloc((void**) &states, PSIZE * sizeof(hiprandState_t));



/*
        Individual *population, *nextPopulation, *swap;
        population = (Individual *) malloc(PSIZE * sizeof(Individual));
        nextPopulation = (Individual *) malloc(PSIZE * sizeof(Individual));
*/
        float *maxFitness;
        maxFitness = (float *) malloc(NGEN * sizeof(float));


        float totalFitness = 0;


        //Create population
        createPopulation<<<PSIZE, 3>>>(population, time(NULL), states);

/*        for(int i = 0; i < PSIZE; i++)
        {
            population[i].fitness = 0;
            population[i].chromossomes = random();
        }*/
    
        //printPop(population, PSIZE, PRINT);
   
        for(int i = 0; i < NGEN; i++)
        {
            //Calculate fitness
            fitness<<<PSIZE, 3>>>(population);

//            totalFitness = fitness(population, PSIZE);
//            std::sort(population, population + PSIZE, comparator);
            thrust::sort(population, population + PSIZE, comparator);
            hipMemcpy(&maxFitness[i], &(population[0].fitness), sizeof(float), hipMemcpyDeviceToHost);
            //maxFitness[i] = population[0].fitness;
            totalFitness = thrust::reduce(population, population + PSIZE, population[0], individualSum).fitness;

            
            reproduce<<<PSIZE, 3>>>(population, nextPopulation, PSIZE, totalFitness, states);
                
            swap = population;
            population = nextPopulation;
            nextPopulation = swap;
        }
/*
        fitness(population, PSIZE);
        std::sort(population, population + PSIZE, comparator);

        //printf("\n");
        //printPop(population, PSIZE, PRINT);
        
        //printf("%f;%x", population[0].fitness, population[0].chromossomes);
        //printf("\n");
     */   
        end = clock();
        hipFree(population);
        hipFree(nextPopulation);
        hipFree(states);

        if(PRINT != 0)
        {   
            printf("Gen\tFitness\n");
            for(int i = 0; i < NGEN; i++)
            {
                printf("%d\t%f\n", i+1, maxFitness[i]);
            }
        }

        free(maxFitness);

        printf("\nT total(us)\t\tT geração(us)\n");
        double cpu_time_used = 1000000 * ((double) (end - start)) / CLOCKS_PER_SEC;
        printf("%f\t\t%f\n\n", cpu_time_used, cpu_time_used/NGEN);
        Ttotal += cpu_time_used;
    }

    printf("\nAvg T total(us)\t\tAvg T geração(us)\n");
    printf("%f\t\t%f\n", Ttotal/NIT, Ttotal/(NIT*NGEN));
    
return 0;
}
