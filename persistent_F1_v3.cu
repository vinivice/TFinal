#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<algorithm>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h>
#include <hip/hip_cooperative_groups.h>


//#define PSIZE 10
//#define NGEN 500000
#define MUT_PROB 0.05
#define TESTE 256

struct Individual 
{
    float fitness;
    unsigned int chromossomes;
};

__device__ bool comparator (Individual i, Individual j)
{
    return (i.fitness > j.fitness);
}

void printPop(Individual *population, int popSize, int print)
{
    if(print != 0)
    {
        for(int i = 0; i < popSize; i++)
        {
            printf("%f - ", population[i].fitness);
        }
    }
}

__device__ int lock = 0;

/*__global__ void lockBlocks(int goal)
{
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    __syncthreads();
    if(threadIdx.x == 0)
    {
        atomicAdd(&lock, 1);
        while(lock != goal)
        {
            printf("AA");
           if(id == 0 && lock == numBlocks)
            {
                printf("BB");
            }
        }
        lock = 0;
    }
    printf("\nlock: %d\tBlocks: %d\tid: %d", lock, goal, id);
    __syncthreads();

}*/


__global__ void persistentThreads(int popSize, int NGEN, float *maxFitness, unsigned int seed, Individual *population, int numBlocks)
{
    grid_group grid = this_grid();
    printf("HERE");
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    Individual child;
    hiprandState_t state;
    hiprand_init(seed, id, 0, &state);

    __shared__ float totalFitness;
    //extern __shared__ Individual population[];


    for(int i = id; i < popSize; i+=blockDim.x * numBlocks)
    {
        //Create population
        population[i].fitness = 0;
        population[i].chromossomes = hiprand(&state);
    }
    grid.sync();

    printf("\nlock: %d\tBlocks: %d\tid: %d", lock, numBlocks, id);
    
    for(int g = 0; g < NGEN; g++)
    {
        if(id == 0)
        {
            totalFitness = 0;
        }
        grid.sync();

        for(int i = id; i < popSize; i+=blockDim.x * numBlocks)
        {
            //Calculate fitness
            
            unsigned int mask = 0x3FF;
            float a = 0, b = 0, c = 0;
            a = population[i].chromossomes & mask;
            b = (population[i].chromossomes & (mask << 10)) >> 10;
            c = (population[i].chromossomes & (mask << 20)) >> 20;

            a = (a - 512)/100.0;
            b = (b - 512)/100.0;
            c = (c - 512)/100.0;

            population[i].fitness = 1.0 / (1 + a*a + b*b + c*c);
            atomicAdd(&totalFitness, population[i].fitness);
        }
        grid.sync();

        if(id == 0)
        {
            thrust::sort(population, population + popSize, comparator);
            maxFitness[g] = population[0].fitness;
        }
        grid.sync();

        
        float localTotalFitness = totalFitness;

        for(int i = id; i < popSize; i+=blockDim.x * numBlocks)
        {

            Individual parents[2];
            int temp = -1;

            //Selection
            for(int j = 0; j < 2; j++)
            {
                float p = hiprand_uniform(&state) * localTotalFitness;
                float score = 0;

                for(int k = 0; k < popSize; k++)
                {
                    if(k == temp)
                    {
                        continue;
                    }
                    score += population[k].fitness;
                    if(p < score)
                    {
                        parents[j] = population[k];
                        localTotalFitness -= population[k].fitness;
                        temp = k;
                        break;
                    }
                }
            }

            //Crossover
            unsigned char cutPoint = hiprand(&state) % 31;
            unsigned int mask1 = 0xffffffff << cutPoint; 
            unsigned int mask2 = 0xffffffff >> (32 - cutPoint);
            child.fitness = 0;
            child.chromossomes = (parents[0].chromossomes & mask1) + (parents[1].chromossomes & mask2);
 
            //Mutation
            float mutation = hiprand_uniform(&state);
            if(mutation < MUT_PROB)
            {
                unsigned char mutPoint = hiprand(&state) % 30;
                child.chromossomes ^= 1 << mutPoint;
            }
        }
        grid.sync();

        
        if(id == 0)
        {
            child = population[0];
        }

        for(int i = id; i < popSize; i+=blockDim.x)
        {
            population[i] = child;
        }
        grid.sync();

        
    }
}

int main(int argc, char *argv[ ]) 
{

    using namespace cooperative_groups;

    int PSIZE, NGEN, NIT, PRINT;
    double Ttotal = 0;
    if(argc < 5)
    {
        printf("Uso %s <POP_SIZE> <N_GEN> <N_ITERACOES> <PRINT>\n", argv[0]);
        return 1;
    }
    else
    {
        PSIZE = atoi(argv[1]);
        NGEN = atoi(argv[2]);
        NIT = atoi(argv[3]);
        PRINT = atoi(argv[4]);
    }

    for(int it = 0; it < NIT; it++)
    {   
        clock_t start, end;
        start = clock();

        float *maxFitness, *cpu_maxFitness;
        hipMalloc((void**) &maxFitness, NGEN * sizeof(float));
        cpu_maxFitness = (float *) malloc(NGEN * sizeof(float));

        Individual *population;
        hipMalloc((void**) &population, PSIZE * sizeof(Individual));

 //       int *lock;
   //     hipMalloc((void**) &lock, sizeof(int));

        int numBlocks = 2;
        int numThreads = 16;


 //       lockBlocks<<<numBlocks, 512>>>(numBlocks);


        hipDevice_t dev;
        hipDeviceGet(&dev,0); 

        hipDeviceProp_t deviceProp;
        int numBlocksPerSm;
        hipGetDeviceProperties(&deviceProp, dev);
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, persistentThreads, numThreads, 0);

        time_t t = time(NULL);
        void **args = ((void **) malloc(6 * sizeof(void *)));
        args[0] = &PSIZE;
        args[1] = &NGEN;
        args[2] = &maxFitness;
        args[3] = &t;
        args[4] = &population;
        args[5] = &numBlocks;

        hipLaunchCooperativeKernel((void*)persistentThreads, deviceProp.multiProcessorCount*numBlocksPerSm, numThreads, args);


 //       persistentThreads<<<numBlocks, 32>>>(PSIZE, NGEN, maxFitness, time(NULL), population, numBlocks);
        //persistentThreads<<<1, min(PSIZE, 1024)>>>(PSIZE, NGEN, maxFitness, time(NULL), population);

        hipMemcpy(cpu_maxFitness, maxFitness, NGEN * sizeof(float), hipMemcpyDeviceToHost);

        end = clock();
        //hipFree(maxFitness);
        if(PRINT != 0)
        {
            printf("Gen\tFitness\n");
            for(int i = 0; i < NGEN; i++)
            {
                printf("%d\t%f\n", i, cpu_maxFitness[i]);
            }
        }
        free(cpu_maxFitness);


        printf("\nT total(us)\t\tT geração(us)\n");
        double cpu_time_used = 1000000 * ((double) (end - start)) / CLOCKS_PER_SEC;
        printf("%f\t\t%f\n\n", cpu_time_used, cpu_time_used/NGEN);
        Ttotal += cpu_time_used;

    }

    printf("\nAvg T total(us)\t\tAvg T geração(us)\n");
    printf("%f\t\t%f\n", Ttotal/NIT, Ttotal/(NIT*NGEN));
    
return 0;
}

